#include "hip/hip_runtime.h"
#ifdef GPU
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#endif
#include <iostream>
#include <vector>
#include <complex>
#include <string>
#include <cstring>
#include <stdio.h>
//#include <bits/stdc++.h>
//#include <unistd.h>
#include <cmath>
#include <cassert>
#include <chrono>
#include "io.hpp"
#include "suqa.cuh"
#include "system.cuh"
#include "Rand.hpp"


using namespace std;

#ifdef GPU
#define NUM_THREADS 128
#define MAXBLOCKS 65535
uint suqa::threads;
uint suqa::blocks;
hipStream_t suqa::stream1, suqa::stream2;
#endif

const int Dim=6;

void self_plaquette(const bmReg& qr0, const bmReg& qr1, const bmReg& qr2, const bmReg& qr3);

int main(int argc, char** argv){
    if(argc<5){
       printf("usage: %s <g_beta> <total_steps> <trotter_stepsize> <outfile>\n",argv[0]); 
       exit(1);
    }
   // g_beta = stod(argv[1]); // (extern) def in src/system.cu
    int total_steps = atoi(argv[2]);
    double trotter_stepsize = stod(argv[3]);
    string outfilename = argv[4];

    //printf("arguments:\n g_beta = %.16lg\n total_steps = %d\n trotter_stepsize = %.16lg\n outfile = %s\n", g_beta, total_steps, trotter_stepsize, outfilename.c_str());

#ifdef GPU
    suqa::threads = NUM_THREADS;
    suqa::blocks = (Dim+suqa::threads-1)/suqa::threads;
    if(suqa::blocks>MAXBLOCKS) suqa::blocks=MAXBLOCKS;
    printf("blocks: %u, threads: %u\n",suqa::blocks, suqa::threads);
#endif

//    ComplexVec state;
  
    suqa::allocate_state(Dim);

    pcg rangen;
    rangen.set_seed(time(NULL));
    rangen.randint(0,3);

    suqa::setup(Dim);
    init_state();

    FILE * outfile;

    DEBUG_CALL(printf("initial state:\n"));
    DEBUG_READ_STATE(suqa::state);

    for(uint ii=0; ii<=(uint)total_steps; ++ii){
        double t = ii*trotter_stepsize;
        printf("time %.16lg\n", t);

//        for(uint hit=0; hit<(uint)num_hits; ++hit){
//            printf("\thit %u\n", hit);
//            init_state(state, Dim, g_beta);
//            evolution(state, t, ii);
//            self_plaquette(state, bm_qlink1, bm_qlink0, bm_qlink2, bm_qlink0);
//            std::vector<uint> c(3);
//            suqa::measure_qbit(state,bm_qlink1[0],c[0],rangen.doub());
//            suqa::measure_qbit(state,bm_qlink1[1],c[1],rangen.doub());
//            suqa::measure_qbit(state,bm_qlink1[2],c[2],rangen.doub());
//            uint plaq_idx = 4*c[2]+2*c[1]+c[0];
//            double plaq_tmp = (plaq_idx==0)? 2.0 : ((plaq_idx==2)? -2.0 : 0.0);
//            plaq_val += plaq_tmp;
//            plaq_val_std += plaq_tmp*plaq_tmp;
//        }
//        plaq_val /=(double)num_hits;
//        plaq_val_std = sqrt((plaq_val_std/(double)num_hits - plaq_val*plaq_val)/(double)(num_hits-1));
//        fprintf(outfile, "%.16lg %d %.16lg %.16lg\n", t, num_hits, plaq_val, plaq_val_std);

        init_state();
		suqa::apply_h(bm_spin[rangen.randint(0,3)]);
        evolution(t, ii);
        printf("random number= %d\n", rangen.randint(0,3));

	//suqa::apply_h(state,  bm_spin[rangen.randint(0,3)]);
	
        double p000=0, p100=0, p010=0, p110=0, p001=0, p101=0, p011=0, p111=0;
        suqa::prob_filter(bm_spin, {0U,0U,0U}, p000);
        suqa::prob_filter(bm_spin, {1U,0U,0U}, p100);
        suqa::prob_filter(bm_spin, {0U,1U,0U}, p010);
        suqa::prob_filter(bm_spin, {1U,1U,0U}, p110);
        suqa::prob_filter(bm_spin, {0U,0U,1U}, p001);
        suqa::prob_filter(bm_spin, {1U,0U,1U}, p101);
        suqa::prob_filter(bm_spin, {0U,1U,1U}, p011);
        suqa::prob_filter(bm_spin, {1U,1U,1U}, p111);
        printf("p000 = %.12lg; p100 = %.12lg\n", p000, p100);
        outfile = fopen(outfilename.c_str(), "a");
        fprintf(outfile, "%.12lg %.12lg %.12lg %.12lg %.12lg %.12lg %.12lg %.12lg %.12lg\n", t, p000,p100, p010,p110, p001,p101, p011,p111);

        fclose(outfile);
    }
    

    suqa::clear();
    
    suqa::deallocate_state();


    return 0;
}
