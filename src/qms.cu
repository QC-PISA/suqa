#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <complex>
#include <string>
#include <cstring>
#include <stdio.h>
//#include <bits/stdc++.h>
//#include <unistd.h>
#include <cmath>
#include <cassert>
#include "Rand.hpp"
#include <chrono>
#ifdef GPU
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#endif
#include "io.hpp"
#include "parser_qms.hpp"
#include "suqa.cuh"
#include "system.cuh"
#include "qms.cuh"

using namespace std;



// simulation parameters
double beta;
double h;
int thermalization;

// defined in src/system.cu
void init_state();

arg_list args;

#ifdef GATECOUNT
GateCounter gctr_global("global");
GateCounter gctr_metrostep("metro step");
GateCounter gctr_sample("sample");
GateCounter gctr_measure("measure");
GateCounter gctr_reverse("reverse");
#endif

void save_measures(string outfilename){
    FILE * fil = fopen(outfilename.c_str(), "a");
    for(uint ei = 0; ei < qms::E_measures.size(); ++ei){
        fprintf(fil, "%.16lg %.16lg\n", qms::E_measures[ei], qms::X_measures[ei]);
    }
    fclose(fil);
    qms::E_measures.clear();
    qms::X_measures.clear();
}

int main(int argc, char** argv){
    if(argc < 7){
        printf("usage: %s <beta> <metro steps> <reset each> <num syst qbits> <num ene qbits> <output file path> [--max-reverse <max reverse attempts> (20)] [--seed <seed> (random)] [--ene-min <min energy> (0.0)] [--ene-max <max energy> (1.0)] [--PE-steps <steps of PE evolution> (10)] [--thermalization <steps> (100)] [--record-reverse]\n", argv[0]);
        exit(1);
    }

    parse_arguments(args, argc, argv);

    beta = args.beta;
//    g_beta = args.g_beta; // defined as extern in system.cuh
    thermalization = args.thermalization;
    qms::metro_steps = (uint)args.metro_steps;
    qms::reset_each = (uint)args.reset_each;
    qms::syst_qbits = (uint)args.syst_qbits;
    qms::ene_qbits = (uint)args.ene_qbits;
    string outfilename(args.outfile);
    qms::max_reverse_attempts = (uint)args.max_reverse_attempts;
    qms::n_phase_estimation = args.pe_steps;
    qms::record_reverse= args.record_reverse;
    qms::iseed = args.seed;
    if(qms::iseed>0)
        qms::rangen.set_seed(qms::iseed);
    
    qms::iseed = qms::rangen.get_seed();

    qms::nqubits = qms::syst_qbits + 2*qms::ene_qbits + 1;
    qms::Dim = (1U << qms::nqubits);
    qms::ene_levels = (1U << qms::ene_qbits);
    qms::syst_levels = (1U << qms::syst_qbits);

    qms::t_PE_shift = args.ene_min;
    qms::t_PE_factor = (qms::ene_levels-1)/(double)(qms::ene_levels*(args.ene_max-args.ene_min)); 
    qms::t_phase_estimation = qms::t_PE_factor*8.*atan(1.0); // 2*pi*t_PE_factor

    
    // Banner
    suqa::print_banner();
    cout<<"arguments:\n"<<args<<endl;

    auto t_start = std::chrono::high_resolution_clock::now();

    // Initialization of utilities
    suqa::setup(qms::nqubits);
    qms::setup(beta);

#ifdef GATECOUNT
    suqa::gatecounters.add_counter(&gctr_global);
    suqa::gatecounters.add_counter(&gctr_metrostep);
    suqa::gatecounters.add_counter(&gctr_sample);
    suqa::gatecounters.add_counter(&gctr_measure);
    suqa::gatecounters.add_counter(&gctr_reverse);

    gctr_global.new_record();
    gctr_sample.new_record();
#endif


    // Initialization:
    // known eigenstate of the system (see src/system.cu)
    
    DEBUG_CALL(cout<<"Preinitial state: "<<endl);
    DEBUG_READ_STATE();
    init_state();
    DEBUG_CALL(cout<<"Initial state: "<<endl);
    DEBUG_READ_STATE();


    //TODO: make it an args option?
    uint perc_mstep = (qms::metro_steps+19)/20; // batched saves
    
    uint count_accepted = 0U;
    if(!file_exists(outfilename.c_str())){
        FILE * fil = fopen(outfilename.c_str(), "w");
        fprintf(fil, "# E A\n");
        fclose(fil);
    }

    bool take_measure;
    uint s0 = 0U;
    //TODO: change metro_steps into actual measures sampled?
    for(uint s = 0U; s < qms::metro_steps; ++s){
        DEBUG_CALL(cout<<"metro step: "<<s<<endl);
        take_measure = (s>s0+(uint)thermalization and (s-s0)%qms::reset_each ==0U);
        int ret = qms::metro_step(take_measure);

        if(ret<0){ // failed rethermalization, reinitialize state
            init_state();
            //ensure new rethermalization
            s0 = s+1; 
        }
        if(ret==1 or ret==2){
            count_accepted++;
        }
        if(s%perc_mstep==0){
            cout<<"iteration: "<<s<<"/"<<qms::metro_steps<<endl;
            save_measures(outfilename);
        }
    }

    cout<<endl;
    printf("\n\tacceptance: %3.2lg%%\n",(count_accepted/static_cast<double>(qms::metro_steps))*100.0);


    qms::clear();
    suqa::clear();

    cout<<"\nall fine :)\n"<<endl;



    if(qms::record_reverse){
        FILE * fil_rev = fopen((outfilename+"_revcounts").c_str(), "w");

        for(uint i = 0; i < qms::reverse_counters.size(); ++i){
            fprintf(fil_rev, "%d %d\n", i, static_cast<int>(qms::reverse_counters[i]));
        }
        fclose(fil_rev);
    }

    cout<<"\n\tSuqa!\n"<<endl;

    auto t_end = std::chrono::high_resolution_clock::now();
    double secs_passed = (1./1000.)*std::chrono::duration<double, std::milli>(t_end-t_start).count();
	cout<<"All [DONE] in "<<secs_passed<<" seconds"<<endl;

    return 0;
}
