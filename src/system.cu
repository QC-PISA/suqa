#include "hip/hip_runtime.h"
#include "system.cuh"

/* d4 gauge theory - two plaquettes
 
   link state 3 qubits
   system state: 4 links -> 12 qubits
   +1 ancillary qubit

 */

double g_beta;

__inline__ double f1(double b){
    return log((3+cosh(2.*b))/(2*sinh(b)*sinh(b)));
}

__inline__ double f2(double b){
    return -log(tanh(b));
}

void init_state(){
    suqa::init_state();

    suqa::apply_h(bm_qlink0[0]);
    suqa::apply_cx(bm_qlink0[0], bm_qlink3[0]);
    suqa::apply_h(bm_qlink0[1]);
    suqa::apply_cx(bm_qlink0[1], bm_qlink3[1]);
    suqa::apply_h(bm_qlink0[2]);
    suqa::apply_cx(bm_qlink0[2], bm_qlink3[2]);
    suqa::apply_mcx({bm_qlink3[0], bm_qlink3[2]}, {0U,1U}, bm_qlink3[1]);
}


/* Quantum evolutor of the state */

void inversion(const bmReg& q){
    suqa::apply_mcx({q[0],q[2]},{1U,0U},q[1]); 
}

void left_multiplication(const bmReg& qr1, const bmReg& qr2){
    suqa::apply_cx(qr1[1], qr2[1]);
    suqa::apply_mcx({qr1[0], qr2[0]}, qr2[1]);
    suqa::apply_cx(qr1[0], qr2[0]);
    suqa::apply_mcx({qr1[0], qr2[2]}, qr2[1]);
    suqa::apply_cx(qr1[2], qr2[2]);
}

void self_plaquette(const bmReg& qr0, const bmReg& qr1, const bmReg& qr2, const bmReg& qr3){
    inversion(qr1);
    left_multiplication(qr1, qr0);
    inversion(qr1);
    inversion(qr2);
    left_multiplication(qr2, qr0);
    inversion(qr2);
    left_multiplication(qr3, qr0);
}

void inverse_self_plaquette(const bmReg& qr0, const bmReg& qr1, const bmReg& qr2, const bmReg& qr3){
    inversion(qr3);
    left_multiplication(qr3, qr0);
    inversion(qr3);
    left_multiplication(qr2, qr0);
    left_multiplication(qr1, qr0);
}

void cphases(uint qaux, uint q0b, double alpha1, double alpha2){
    suqa::apply_cx(qaux, q0b);
    suqa::apply_cu1(q0b, qaux, alpha1, 1U);
    suqa::apply_cx(qaux, q0b);
    suqa::apply_cu1(q0b, qaux, alpha2, 1U);
}

void self_trace_operator(const bmReg& qr, const uint& qaux, double th){
    suqa::apply_mcx({qr[0],qr[2]}, {0U,0U}, qaux); 
    cphases(qaux, qr[1], th, -th);
    suqa::apply_mcx({qr[0],qr[2]}, {0U,0U}, qaux); 
}

void fourier_transf_d4(const bmReg& qr){
    suqa::apply_cx(qr[2], qr[0]);
    suqa::apply_cx(qr[0], qr[2]);
    suqa::apply_tdg(qr[2]);
    suqa::apply_tdg(qr[2]);
    suqa::apply_cx(qr[1], qr[2]);
    suqa::apply_h(qr[0]);
    suqa::apply_h(qr[1]);
    suqa::apply_h(qr[2]);
    suqa::apply_t(qr[1]);
    suqa::apply_tdg(qr[2]);
    suqa::apply_cx(qr[1], qr[2]);
    suqa::apply_cx(qr[0], qr[1]);
    suqa::apply_h(qr[1]);
    suqa::apply_t(qr[1]);
    suqa::apply_t(qr[1]);
    suqa::apply_h(qr[1]);
}


void inverse_fourier_transf_d4(const bmReg& qr){
    suqa::apply_h(qr[1]);
    suqa::apply_tdg(qr[1]);
    suqa::apply_tdg(qr[1]);
    suqa::apply_h(qr[1]);
    suqa::apply_cx(qr[0], qr[1]);
    suqa::apply_cx(qr[1], qr[2]);
    suqa::apply_t(qr[2]);
    suqa::apply_tdg(qr[1]);
    suqa::apply_h(qr[0]);
    suqa::apply_h(qr[1]);
    suqa::apply_h(qr[2]);
    suqa::apply_cx(qr[1], qr[2]);
    suqa::apply_t(qr[2]);
    suqa::apply_t(qr[2]);
    suqa::apply_cx(qr[0], qr[2]);
    suqa::apply_cx(qr[2], qr[0]);
}

void momentum_phase(const bmReg& qr, const uint& qaux, double th1, double th2){
    suqa::apply_mcx(qr, {0U,0U,0U}, qaux);
    DEBUG_CALL(printf("\tafter suqa::apply_mcx(qr, {0U,0U,0U}, qaux)\n"));
    DEBUG_READ_STATE();
    suqa::apply_cx(qaux, qr[2]);
    suqa::apply_cu1(qaux, qr[2], th1);
    suqa::apply_cx(qaux, qr[2]);
    DEBUG_CALL(printf("\tafter suqa::apply_cu1(qaux, qr[2], th1, 0U)\n"));
    DEBUG_READ_STATE();
    suqa::apply_u1(qr[2], th2);
    DEBUG_CALL(printf("\tafter suqa::apply_u1(qr[2], th2)\n"));
    DEBUG_READ_STATE();
    suqa::apply_mcx(qr, {0U,0U,0U}, qaux);
    DEBUG_CALL(printf("\tafter suqa::apply_mcx(qr, {0U,0U,0U}, qaux)\n"));
    DEBUG_READ_STATE();
}

void evolution(const double& t, const int& n){
    const double dt = t/(double)n;

    const double theta1 = dt*f1(g_beta);
    const double theta2 = dt*f2(g_beta);
    const double theta = 2*dt*g_beta;
//    printf("g_beta = %.16lg, dt = %.16lg, thetas: %.16lg %.16lg %.16lg\n", g_beta, dt, theta1, theta2, theta);

    for(uint ti=0; ti<(uint)n; ++ti){
        self_plaquette(bm_qlink1, bm_qlink0, bm_qlink2, bm_qlink0);
        DEBUG_CALL(printf("after self_plaquette()\n"));
        DEBUG_READ_STATE();
        self_trace_operator(bm_qlink1, bm_qaux[0], theta);
        DEBUG_CALL(printf("after self_trace_operator()\n"));
        DEBUG_READ_STATE();
        inverse_self_plaquette(bm_qlink1, bm_qlink0, bm_qlink2, bm_qlink0);
        DEBUG_CALL(printf("after inverse_self_plaquette()\n"));
        DEBUG_READ_STATE();

        self_plaquette(bm_qlink2, bm_qlink3, bm_qlink1, bm_qlink3);
        DEBUG_CALL(printf("after self_plaquette()\n"));
        DEBUG_READ_STATE();
        self_trace_operator(bm_qlink2, bm_qaux[0], theta);
        DEBUG_CALL(printf("after self_trace_operator()\n"));
        DEBUG_READ_STATE();
        inverse_self_plaquette(bm_qlink2, bm_qlink3, bm_qlink1, bm_qlink3);
        DEBUG_CALL(printf("after inverse_self_plaquette()\n"));
        DEBUG_READ_STATE();

        fourier_transf_d4(bm_qlink0);
        DEBUG_CALL(printf("after fourier_transf_d4(bm_qlink0)\n"));
        DEBUG_READ_STATE();
        fourier_transf_d4(bm_qlink1);
        DEBUG_CALL(printf("after fourier_transf_d4(bm_qlink1)\n"));
        DEBUG_READ_STATE();
        fourier_transf_d4(bm_qlink2);
        DEBUG_CALL(printf("after fourier_transf_d4(bm_qlink2)\n"));
        DEBUG_READ_STATE();
        fourier_transf_d4(bm_qlink3);
        DEBUG_CALL(printf("after fourier_transf_d4(bm_qlink3)\n"));
        DEBUG_READ_STATE();

        momentum_phase(bm_qlink0, bm_qaux[0], theta1, theta2);
        DEBUG_CALL(printf("after momentum_phase(bm_qlink0, bm_qaux[0], theta1, theta2)\n"));
        DEBUG_READ_STATE();
        momentum_phase(bm_qlink1, bm_qaux[0], theta1, theta2);
        DEBUG_CALL(printf("after momentum_phase(bm_qlink1, bm_qaux[0], theta1, theta2)\n"));
        DEBUG_READ_STATE();
        momentum_phase(bm_qlink2, bm_qaux[0], theta1, theta2);
        DEBUG_CALL(printf("after momentum_phase(bm_qlink2, bm_qaux[0], theta1, theta2)\n"));
        DEBUG_READ_STATE();
        momentum_phase(bm_qlink3, bm_qaux[0], theta1, theta2);
        DEBUG_CALL(printf("after momentum_phase(bm_qlink3, bm_qaux[0], theta1, theta2)\n"));
        DEBUG_READ_STATE();


        inverse_fourier_transf_d4(bm_qlink3);
        DEBUG_CALL(printf("after inverse_fourier_transf_d4(bm_qlink3)\n"));
        DEBUG_READ_STATE();
        inverse_fourier_transf_d4(bm_qlink2);
        DEBUG_CALL(printf("after inverse_fourier_transf_d4(bm_qlink2)\n"));
        DEBUG_READ_STATE();
        inverse_fourier_transf_d4(bm_qlink1);
        DEBUG_CALL(printf("after inverse_fourier_transf_d4(bm_qlink1)\n"));
        DEBUG_READ_STATE();
        inverse_fourier_transf_d4(bm_qlink0);
        DEBUG_CALL(printf("after inverse_fourier_transf_d4(bm_qlink0)\n"));
        DEBUG_READ_STATE();
    }
}



// qsa specifics
void qsa_init_state(){
    //TODO: implement
    throw std::runtime_error("ERROR: qsa_init_state() unimplemented!\n");
//    suqa::init_state();
//    suqa::apply_h(bm_spin[0]);
//    suqa::apply_h(bm_spin[1]);
//    suqa::apply_h(bm_spin[2]);
//    suqa::apply_cx(bm_spin[0], bm_spin_tilde[0]);
//    suqa::apply_cx(bm_spin[1], bm_spin_tilde[1]);
//    suqa::apply_cx(bm_spin[2], bm_spin_tilde[2]);
}

void evolution_szegedy(const double& t, const int& n){
    (void)t,(void)n;
    //TODO: implement
    throw std::runtime_error("ERROR: evolution_szegedy() unimplemented!\n");
//    (void)n;
//      DEBUG_CALL(std::cout<<"before evolution_szegedy()"<<std::endl);
//      DEBUG_READ_STATE();
//      DEBUG_CALL(std::cout<<"apply evolution_szegedy()"<<std::endl);
//    for (uint i = 0; i < 3; i++) {
//      suqa::apply_pauli_TP_rotation({bm_spin_tilde[(0+i)%3],bm_spin_tilde[(1+i)%3]}, {PAULI_X,PAULI_X}, -t);
//      DEBUG_CALL(std::cout<<"apply pauli 1 it "<<i<<std::endl);
//      DEBUG_READ_STATE();
//      suqa::apply_pauli_TP_rotation({bm_spin[(0+i)%3],bm_spin[(1+i)%3]}, {PAULI_X,PAULI_X}, t);
//      DEBUG_CALL(std::cout<<"apply pauli 2 it "<<i<<std::endl);
//      DEBUG_READ_STATE();
//    }
}

void evolution_measure(const double& t, const int& n){
    (void)t,(void)n;
    //TODO: implement
    throw std::runtime_error("ERROR: evolution_measure() unimplemented!\n");
}

void evolution_tracing(const double& t, const int& n){
    (void)t,(void)n;
    //TODO: implement
    throw std::runtime_error("ERROR: evolution_tracing() unimplemented!\n");
//    (void)n;
//  for (uint i = 0; i < 3; i++) {
//    suqa::apply_pauli_TP_rotation({bm_spin_tilde[(0+i)%3],bm_spin_tilde[(1+i)%3]}, {PAULI_X,PAULI_X}, -t);
//  }
//
}

/* Measure facilities */
const uint op_bits = 3; // 2^op_bits is the number of eigenvalues for the observable
const bmReg bm_op = bm_qlink1; // where the measure has to be taken
const std::vector<double> op_vals = {2.0,0.0,-2.0, 0.0,0.0,0.0,0.0,0.0}; // eigvals

 
// change basis to the observable basis somewhere in the system registers
void apply_measure_rotation(){
    self_plaquette(bm_qlink1, bm_qlink0, bm_qlink2, bm_qlink0);
}

// inverse of the above function
void apply_measure_antirotation(){
    inverse_self_plaquette(bm_qlink1, bm_qlink0, bm_qlink2, bm_qlink0);
}

// map the classical measure recorded in creg_vals
// to the corresponding value of the observable;
// there is no need to change it
double get_meas_opvals(const uint& creg_vals){
    return op_vals[creg_vals];
}

// actually perform the measure
// there is no need to change it
double measure_X(pcg& rgen){
    std::vector<uint> classics(op_bits);
    
    apply_measure_rotation();

    std::vector<double> rdoubs(op_bits);
    for(auto& el : rdoubs){
        el = rgen.doub();
    }
    suqa::measure_qbits(bm_op, classics, rdoubs);

    apply_measure_antirotation();

    uint meas = 0U;
    for(uint i=0; i<op_bits; ++i){
        meas |= (classics[i] << i);
    }

    return get_meas_opvals(meas);
}

/* Moves facilities */

std::vector<double> C_weigthsums = {1./3, 2./3, 1.0};

void apply_C(const uint &Ci){
    (void)Ci;
    //TODO: implement
    throw std::runtime_error("ERROR: apply_C() unimplemented!\n");
}

void apply_C_inverse(const uint &Ci){
    (void)Ci;
    //TODO: implement
    throw std::runtime_error("ERROR: apply_C_inverse() unimplemented!\n");
}

void qsa_apply_C(const uint &Ci){
    (void)Ci;
    //TODO: implement
    throw std::runtime_error("ERROR: qsa_apply_C() unimplemented!\n");
//  suqa::apply_h(bm_spin_tilde[Ci]);
// suqa::apply_h(state,bm_spin_tilde[(Ci+1)%3]);


  // suqa::apply_h(state,bm_spin_tilde);
}

void qsa_apply_C_inverse(const uint &Ci){
    (void)Ci;
    //TODO: implement
    throw std::runtime_error("ERROR: qsa_apply_C() unimplemented!\n");
//  if(Ci>2) throw std::runtime_error("ERROR: wrong move selection");
//  //suqa::apply_h(state,bm_spin_tilde);
//  //suqa::apply_h(state,bm_spin_tilde[(Ci+1)%3]);
//  suqa::apply_h(bm_spin_tilde[Ci]);
}

std::vector<double> get_C_weigthsums(){ return C_weigthsums; }

