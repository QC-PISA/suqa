#ifdef GPU
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#endif
#include <iostream>
#include <vector>
#include <complex>
#include <string>
#include <cstring>
#include <stdio.h>
#include <cmath>
#include <cassert>
#include <chrono>
#include "io.hpp"
#include "suqa.cuh"



int main(int argc, char** argv) {

	suqa::setup(5);
	suqa::init_state();

	DEBUG_CALL(printf("Initial state:\n"));
	DEBUG_READ_STATE(suqa::state);

	suqa::apply_x(3);
	DEBUG_CALL(printf("After apply_x(3):\n"));
	DEBUG_READ_STATE(suqa::state);

	suqa::apply_x(3);
	DEBUG_CALL(printf("After apply_x(3):\n"));
	DEBUG_READ_STATE(suqa::state);

	suqa::apply_h(1);
	DEBUG_CALL(printf("After apply_h(1):\n"));
	DEBUG_READ_STATE(suqa::state);

	suqa::apply_x(3);
	DEBUG_CALL(printf("After apply_x(3):\n"));
	DEBUG_READ_STATE(suqa::state);

	suqa::apply_h(1);
	DEBUG_CALL(printf("After apply_h(1):\n"));
	DEBUG_READ_STATE(suqa::state);

	suqa::apply_cx(3,0);
	DEBUG_CALL(printf("After apply_cx(3,0):\n"));
	DEBUG_READ_STATE(suqa::state);

	suqa::apply_cx(3,0);
	DEBUG_CALL(printf("After apply_cx(3,0):\n"));
	DEBUG_READ_STATE(suqa::state);

	suqa::apply_h(0);
	DEBUG_CALL(printf("After apply_h(0):\n"));
	DEBUG_READ_STATE(suqa::state);

	suqa::apply_h(1);
	DEBUG_CALL(printf("After apply_h(1):\n"));
	DEBUG_READ_STATE(suqa::state);

	suqa::apply_h({ 2, 4 });
	DEBUG_CALL(printf("After apply_h({2,4}):\n"));
	DEBUG_READ_STATE(suqa::state);

	suqa::apply_cx(0,4);
	DEBUG_CALL(printf("After apply_cx(0,4):\n"));
	DEBUG_READ_STATE(suqa::state);

	suqa::apply_u1(3,M_PI/3.0);
	DEBUG_CALL(printf("After apply_u1(3,M_PI/3.0):\n"));
	DEBUG_READ_STATE(suqa::state);

	suqa::clear();

	return 0;
}
