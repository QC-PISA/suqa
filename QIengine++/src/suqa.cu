#include "hip/hip_runtime.h"
#include "suqa.cuh"
//#include "cub/hipcub/hipcub.hpp" 
//#include <thrust/transform_reduce.h>
//#include <thrust/execution_policy.h>
//#include <thrust/functional.h>

#if !defined(NDEBUG)
double *host_state_re, *host_state_im;
#endif

double *host_partial_ret, *dev_partial_ret;

// global control mask:
// it applies every next operation 
// using it as condition (the user should make sure
// to use it only for operations not involving it)
uint suqa::gc_mask;

void suqa::activate_gc_mask(const bmReg& q_controls){
    suqa::gc_mask=0U;
    for(const auto& q : q_controls)
        suqa::gc_mask |= 1U << q;
}

void suqa::deactivate_gc_mask(){
    suqa::gc_mask=0U;
}

/*XXX: everything is optimizable
 *possible strategies:
 *
 *  * sparse data structure: worse in the worst case, but allows bigger states
 *    and potential speedup (depends on the algorithm)
 *    - caching: only useful with sparse data structures, due to bad scaling
 *      in storage consumption.
 *      Allocating it in constant memory would be pretty (only 64KB allowed, though)
 * 
 *  * grey encoding + shared memory: to improve coalescing
 *
 *  * exploit cpu: instead of caching for example, to precompute masks (e.g. in mcx) 
 */


//TODO: optimize reduce
__global__ void kernel_suqa_vnorm(double *dev_partial_ret_ptr, double *v_re, double *v_im, uint len){
    extern __shared__ double local_ret[];
    uint i =  blockIdx.x*blockDim.x + threadIdx.x;

//    double vj = v_comp[i+(blockDim.x >> 1)];
//    local_ret[tid] =  v_comp[i]*v_comp[i]+vj*vj;
    local_ret[threadIdx.x] = 0.0;
    double tmpval;
    while(i<len){
        tmpval = v_re[i]; 
        local_ret[threadIdx.x] +=  tmpval*tmpval;
        tmpval = v_im[i]; 
        local_ret[threadIdx.x] +=  tmpval*tmpval;
//        if(v_re[i]>0.0)
//            printf("%u %.16lg, %.16lg; loc_ret[%d] = %.16lg\n",i, v_re[i], v_im[i], threadIdx.x, local_ret[threadIdx.x]);
//        tmpval = v_comp[i+blockDim.x]; 
//        local_ret[threadIdx.x] +=  tmpval*tmpval;
        i += gridDim.x*blockDim.x;
//        printf("v[%d] = (%.16lg, %.16lg)\n",i, v_re[i], v_im[i]);
//        printf("local_ret[%d] = %.10lg\n",threadIdx.x, local_ret[threadIdx.x]);

    }
    __syncthreads();

    for(uint s=blockDim.x/2; s>0; s>>=1){
        if(threadIdx.x < s){
            local_ret[threadIdx.x] += local_ret[threadIdx.x+s];
        }
        __syncthreads();
    }
//    if (blockDim.x >= 1024) { if (threadIdx.x < 512) { local_ret[threadIdx.x] += local_ret[threadIdx.x + 512]; } __syncthreads(); }
//    if (blockDim.x >=  512) { if (threadIdx.x < 256) { local_ret[threadIdx.x] += local_ret[threadIdx.x + 256]; } __syncthreads(); }
//    if (blockDim.x >=  256) { if (threadIdx.x < 128) { local_ret[threadIdx.x] += local_ret[threadIdx.x + 128]; } __syncthreads(); }
//    if (blockDim.x >=  128) { if (threadIdx.x <  64) { local_ret[threadIdx.x] += local_ret[threadIdx.x +  64]; } __syncthreads(); }
//
//    if(threadIdx.x<32){
//        if (blockDim.x >= 64) local_ret[threadIdx.x] += local_ret[threadIdx.x + 32];
//        if (blockDim.x >= 32) local_ret[threadIdx.x] += local_ret[threadIdx.x + 16];
//        if (blockDim.x >= 16) local_ret[threadIdx.x] += local_ret[threadIdx.x +  8];
//        if (blockDim.x >=  8) local_ret[threadIdx.x] += local_ret[threadIdx.x +  4];
//        if (blockDim.x >=  4) local_ret[threadIdx.x] += local_ret[threadIdx.x +  2];
//        if (blockDim.x >=  2) local_ret[threadIdx.x] += local_ret[threadIdx.x +  1];
//    }

    if(threadIdx.x==0){
        dev_partial_ret_ptr[blockIdx.x] = local_ret[0];
//        printf("dev_partial_ret_ptr[%d] = %.16lg\n",blockIdx.x,dev_partial_ret_ptr[blockIdx.x]);
    }
}

double suqa::vnorm(const ComplexVec& v){
    
    double ret = 0.0;
//    double *host_partial_ret = new double[suqa::blocks];
    
    kernel_suqa_vnorm<<<suqa::blocks,suqa::threads,suqa::threads*sizeof(double)>>>(dev_partial_ret, v.data_re, v.data_im, v.size());
//    kernel_suqa_vnorm<<<suqa::blocks,suqa::threads,suqa::threads*sizeof(double),suqa::stream1>>>(dev_partial_ret, v.data_re, v.size());
//    kernel_suqa_vnorm<<<suqa::blocks,suqa::threads,suqa::threads*sizeof(double),suqa::stream2>>>(dev_partial_ret+suqa::blocks*sizeof(double),  v.data_im, v.size());
//    hipDeviceSynchronize();

    hipMemcpy(host_partial_ret,dev_partial_ret,suqa::blocks*sizeof(double), hipMemcpyDeviceToHost);
//    hipDeviceSynchronize();
    
    for(uint bid=0; bid<suqa::blocks; ++bid){
        
//        printf("host_partial_ret[%d(/%d)] = %.10lg\n",bid, suqa::blocks,host_partial_ret[bid]);
        ret += host_partial_ret[bid]; 
    } 
    return sqrt(ret);
}

////XXX Other possibility: libraries: thrust and cub

////XXX thrust version
//template<typename T>
//struct Square{
// __host__ __device__ __forceinline__
//  T operator()(const T& a) const {
//    return a*a;
//  }
//};
//
//
//template<typename Iterator, typename T, typename UnaryOperation, typename BinaryOperation, typename Pointer>
//__global__ void reduce_kernel(Iterator first, Iterator last, UnaryOperation unary_op, T init, BinaryOperation binary_op, Pointer result){
//  *result = thrust::transform_reduce(thrust::cuda::par, first, last, unary_op, init, binary_op);
//}

//double *ret_re_im, *d_ret_re_im;
//double suqa::vnorm(const ComplexVec& v){
//    
//    double ret = 0.0;
////    double *host_partial_ret = new double[suqa::blocks];
//    Square<double> unary_op;
//    thrust::plus<double> binary_op;
//
////    ret_re_im[0] = thrust::transform_reduce(thrust::cuda::par.on(stream1) , v.data_re, &v.data_re[v.size()-1], unary_op, 0.0, binary_op);
////    ret_re_im[1] = thrust::transform_reduce(thrust::cuda::par.on(stream2), v.data_im, &v.data_im[v.size()-1], unary_op, 0.0, binary_op);
//
//    reduce_kernel<<<1,1,0,stream1>>>(v.data_re, &v.data_re[v.size()-1], unary_op, 0.0, binary_op, &d_ret_re_im[0]);
//    reduce_kernel<<<1,1,0,stream2>>>(v.data_im, v.data_im+v.size(), unary_op, 0.0, binary_op, &d_ret_re_im[1]);
//    hipMemcpyAsync(ret_re_im+1,d_ret_re_im+1,sizeof(double),hipMemcpyDeviceToHost, stream2);
//    hipMemcpyAsync(ret_re_im,d_ret_re_im,sizeof(double),hipMemcpyDeviceToHost, stream1);
//
//    hipStreamSynchronize(stream1);
//    hipStreamSynchronize(stream2);
//    ret +=ret_re_im[0]+ret_re_im[1];
//    
//
////    printf("ret: %.16lg = %.16lg + %.16lg\n",ret, ret_re, ret_im);
//
//
//    return sqrt(ret);
//}

////XXX  CUB version
// double *ret_re_im, *d_ret_re_im;
// double suqa::vnorm(const ComplexVec& v){
//     
//     double ret = 0.0;
// //    double *host_partial_ret = new double[suqa::blocks];
// 
//     hipcub::TransformInputIterator<double, Square<double>, double*> input_iter_re(v.data_re, Square<double>());
//     hipcub::TransformInputIterator<double, Square<double>, double*> input_iter_im(v.data_im, Square<double>());
// 
// 
// 
//     void     *d_temp_storage = NULL;
//     size_t   temp_storage_bytes = 0;
//     hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, input_iter_re, d_ret_re_im, v.size());
//     // Allocate temporary storage
//     hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
//     // Run sum-reduction
// //    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, input_iter_re, &d_ret_re_im[0], v.size(),stream1);
// //    hipMemcpyAsync(&ret_re_im[0],d_ret_re_im,sizeof(double),hipMemcpyDeviceToHost,stream1);
// //    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, input_iter_im, &d_ret_re_im[1], v.size(),stream2);
// //    hipMemcpyAsync(&ret_re_im[1],&d_ret_re_im[1],sizeof(double),hipMemcpyDeviceToHost,stream2); // synchronous
// //    hipDeviceSynchronize();
//     hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, input_iter_re, d_ret_re_im, v.size(),suqa::stream1);
//     hipStreamSynchronize(stream1);
//     hipMemcpyAsync(&ret_re_im[0],&d_ret_re_im[0],sizeof(double),hipMemcpyDeviceToHost,stream1);
//     hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, input_iter_im, &d_ret_re_im[1], v.size(),suqa::stream2);
//     hipStreamSynchronize(stream2);
//     hipMemcpyAsync(&ret_re_im[1],&d_ret_re_im[1],sizeof(double),hipMemcpyDeviceToHost,stream2); // synchronous
// 
//     hipStreamSynchronize(stream1);
//     ret +=ret_re_im[0];
//     hipStreamSynchronize(stream2);
//     ret +=ret_re_im[1];
// 
//     ret =ret_re_im[0]+ret_re_im[1];
// 
// //    printf("ret: %.16lg = %.16lg + %.16lg\n",ret, ret_re_im[0], ret_re_im[1]);
// 
//     hipFree(d_temp_storage);
// 
//     return sqrt(ret);
// }

//__launch_bounds__(128, 6)
__global__ void kernel_suqa_vnormalize_by(double *v_comp, uint len, double value){
    uint i =  blockIdx.x*blockDim.x + threadIdx.x;
    while(i < len){
        v_comp[i]*=value;
        i += gridDim.x*blockDim.x;
    }
}

void suqa::vnormalize(ComplexVec& v){
    double vec_norm = suqa::vnorm(v);
#ifndef NDEBUG
    std::cout<<"vec_norm = "<<vec_norm<<std::endl;
#endif
    // using the inverse, since division is not built-in in cuda
    kernel_suqa_vnormalize_by<<<suqa::blocks,suqa::threads>>>(v.data, 2*v.size(),1./vec_norm);
//    kernel_suqa_vnormalize_by<<<suqa::blocks,suqa::threads, 0, suqa::stream1>>>(v.data_re, v.size(),1./vec_norm);
//    kernel_suqa_vnormalize_by<<<suqa::blocks,suqa::threads, 0, suqa::stream2>>>(v.data_im, v.size(),1./vec_norm);
    hipDeviceSynchronize();
}



//  X GATE


__global__ 
void kernel_suqa_x(double *const state_re, double *const state_im, uint len, uint q, uint glob_mask){
    int i = blockDim.x*blockIdx.x + threadIdx.x;    
    glob_mask |= (1U <<q);
    while(i<len){
        if((i & glob_mask) == glob_mask){
            uint j = i & ~(1U << q); // j has 0 on q-th digit
            double tmpval = state_re[i];
            state_re[i]=state_re[j];
            state_re[j]=tmpval;
            tmpval = state_im[i];
            state_im[i]=state_im[j];
            state_im[j]=tmpval;
        }
        i+=gridDim.x*blockDim.x;
    }
}


void suqa::apply_x(ComplexVec& state, uint q){
    kernel_suqa_x<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), q, gc_mask);
}  

// no way...
//__global__ 
//void kernel_suqa_mx(Complex *const state, uint len, uint msq, uint mask){
//    // msq here stands for most significant qubit
//    uint i = blockDim.x*blockIdx.x + threadIdx.x;
//    while(i<len){
//        if(i & (1U << msq)){
//            uint j = i & ~(1U << msq); // j has 0 on q-th digit
//            swap_cmpx(&state[i],&state[j]);
//        }
//        i+=gridDim.x*blockDim.x;
//    }
//}


void suqa::apply_x(ComplexVec& state, const bmReg& qs){
    for(const auto& q : qs)
        kernel_suqa_x<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), q, gc_mask);
}  
//void suqa::qi_x(ComplexVec& state, const vector<uint>& qs){
//    for(const auto& q : qs)
//        qi_x(state, q);
//}  

//  HADAMARD GATE

__global__ 
void kernel_suqa_h(double *state_re, double *state_im, uint len, uint q, uint glob_mask){
//    const Complex TWOSQINV_CMPX = make_hipDoubleComplex(TWOSQINV,0.0f);
     
    uint i_0 = blockDim.x*blockIdx.x + threadIdx.x;    
    
    uint loc_mask = glob_mask | (1U << q);
    while(i_0<len){
        if((i_0 & loc_mask) == glob_mask){
            const uint i_1 = i_0 | (1U << q);
            double a_0_re = state_re[i_0];
            double a_1_re = state_re[i_1];
            double a_0_im = state_im[i_0];
            double a_1_im = state_im[i_1];
            
            state_re[i_0]= TWOSQINV*(a_0_re+a_1_re);
            state_re[i_1]= TWOSQINV*(a_0_re-a_1_re);
            state_im[i_0]= TWOSQINV*(a_0_im+a_1_im);
            state_im[i_1]= TWOSQINV*(a_0_im-a_1_im);
        }
        i_0+=gridDim.x*blockDim.x;
    }
}


void suqa::apply_h(ComplexVec& state, uint q){
    kernel_suqa_h<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), q, gc_mask);
    DEBUG_READ_STATE(state);
}  


void suqa::apply_h(ComplexVec& state, const bmReg& qs){
    for(const auto& q : qs){
        kernel_suqa_h<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), q, gc_mask);
    }
}  

//  PI/8 GATES

__global__ 
void kernel_suqa_t(double *state_re, double *state_im, uint len, uint q, uint glob_mask){
//    const Complex TWOSQINV_CMPX = make_hipDoubleComplex(TWOSQINV,0.0f);
     
    uint i_1 = blockDim.x*blockIdx.x + threadIdx.x;    
    glob_mask |= (1U << q);
    while(i_1<len){
        if((i_1 & glob_mask) == glob_mask){
            double a_1_re = state_re[i_1];
            double a_1_im = state_im[i_1];
            
            state_re[i_1]= TWOSQINV*(a_1_re - a_1_im);
            state_im[i_1]= TWOSQINV*(a_1_im + a_1_re);
        }
        i_1+=gridDim.x*blockDim.x;
    }
}

__global__ 
void kernel_suqa_tdg(double *state_re, double *state_im, uint len, uint q, uint glob_mask){
//    const Complex TWOSQINV_CMPX = make_hipDoubleComplex(TWOSQINV,0.0f);
     
    uint i_1 = blockDim.x*blockIdx.x + threadIdx.x;    
    glob_mask |= (1U << q);
    while(i_1<len){
        if((i_1 & glob_mask) == glob_mask){
            double a_1_re = state_re[i_1];
            double a_1_im = state_im[i_1];
            
            state_re[i_1]= TWOSQINV*(a_1_re + a_1_im);
            state_im[i_1]= TWOSQINV*(a_1_im - a_1_re);
        }
        i_1+=gridDim.x*blockDim.x;
    }
}


// T gate (single qubit)
void suqa::apply_t(ComplexVec& state, uint q){
    kernel_suqa_t<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), q, gc_mask);
}  
// T gate (multiple qubits)
void suqa::apply_t(ComplexVec& state, const bmReg& qs){
    for(const auto& q : qs){
        kernel_suqa_t<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), q, gc_mask);
    }
}  

// T^{\dagger} gate (single qubit)
void suqa::apply_tdg(ComplexVec& state, uint q){
    kernel_suqa_tdg<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), q, gc_mask);
}  
// T^{\dagger} gate (multiple qubits)
void suqa::apply_tdg(ComplexVec& state, const bmReg& qs){
    for(const auto& q : qs){
        kernel_suqa_tdg<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), q, gc_mask);
    }
}  

// U1 GATE

__global__ 
void kernel_suqa_u1(double *state_re, double *state_im, uint len, uint q, Complex phase, uint glob_mask){
//    const Complex TWOSQINV_CMPX = make_hipDoubleComplex(TWOSQINV,0.0f);
     
    uint i_1 = blockDim.x*blockIdx.x + threadIdx.x;    
    glob_mask |= (1U << q);
    while(i_1<len){
        if((i_1 & glob_mask) == glob_mask){
            double tmpval = state_re[i_1]; 
            state_re[i_1] = state_re[i_1]*phase.x-state_im[i_1]*phase.y;
            state_im[i_1] = tmpval*phase.y+state_im[i_1]*phase.x;

        }
        i_1+=gridDim.x*blockDim.x;
    }
}


void suqa::apply_u1(ComplexVec& state, uint q, double phase){
    Complex phasec;
    sincos(phase, &phasec.y, &phasec.x);
    kernel_suqa_u1<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), q, phasec, gc_mask);
}


//  CONTROLLED-NOT GATE

__global__ 
void kernel_suqa_mcx(double *const state_re, double *const state_im, uint len, uint control_mask, uint mask_qs, uint q_target){
    int i = blockDim.x*blockIdx.x + threadIdx.x;    
    while(i<len){
        if((i & control_mask) == mask_qs){
            uint j = i & ~(1U << q_target);
            double tmpval = state_re[i];
            state_re[i]=state_re[j];
            state_re[j]=tmpval;
            tmpval = state_im[i];
            state_im[i]=state_im[j];
            state_im[j]=tmpval;
        }
        i+=gridDim.x*blockDim.x;
    }
}


void suqa::apply_cx(ComplexVec& state, const uint& q_control, const uint& q_target, const uint& q_mask){
    uint mask_qs = (1U << q_target) | gc_mask;
    uint mask = mask_qs | (1U << q_control);
    if(q_mask) mask_qs |= (1U << q_control);
    kernel_suqa_mcx<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), mask, mask_qs, q_target);
}  

void suqa::apply_mcx(ComplexVec& state, const bmReg& q_controls, const uint& q_target){
    uint mask = (1U << q_target) | gc_mask;
    for(const auto& q : q_controls)
        mask |= 1U << q;
    kernel_suqa_mcx<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), mask, mask, q_target);
}  


void suqa::apply_mcx(ComplexVec& state, const bmReg& q_controls, const bmReg& q_mask, const uint& q_target){
    uint mask = (1U << q_target) | gc_mask;
    for(const auto& q : q_controls)
        mask |= 1U << q;
    uint mask_qs = (1U << q_target) | gc_mask;
    for(uint k = 0U; k < q_controls.size(); ++k){
        if(q_mask[k]) mask_qs |= 1U << q_controls[k];
    }

    kernel_suqa_mcx<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), mask, mask_qs, q_target);
}  

__global__ 
void kernel_suqa_mcu1(double *const state_re, double *const state_im, uint len, uint control_mask, uint mask_qs, uint q_target, Complex rphase){
    int i = blockDim.x*blockIdx.x + threadIdx.x;    
    while(i<len){
        if((i & control_mask) == mask_qs){
//            uint j = i & ~(1U << q_target);
            double tmpval = state_re[i]; 
            state_re[i] = state_re[i]*rphase.x-state_im[i]*rphase.y;
            state_im[i] = tmpval*rphase.y+state_im[i]*rphase.x;
        }
        i+=gridDim.x*blockDim.x;
    }
}

void suqa::apply_cu1(ComplexVec& state, uint q_control, uint q_target, double phase, uint q_mask){
    uint mask_qs = (1U << q_target) | gc_mask;
    uint mask = mask_qs | (1U << q_control);
    if(q_mask) mask_qs |= (1U << q_control);

    Complex phasec;
    sincos(phase, &phasec.y, &phasec.x);

    kernel_suqa_mcu1<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), mask, mask_qs, q_target, phasec);
}

void suqa::apply_mcu1(ComplexVec& state, const bmReg& q_controls, const bmReg& q_mask, const uint& q_target, double phase){
    uint mask = (1U << q_target) | gc_mask;
    for(const auto& q : q_controls)
        mask |= 1U << q;
    uint mask_qs = (1U << q_target) | gc_mask;
    for(uint k = 0U; k < q_controls.size(); ++k){
        if(q_mask[k]) mask_qs |= 1U << q_controls[k];
    }

    Complex phasec;
    sincos(phase, &phasec.y, &phasec.x);

    kernel_suqa_mcu1<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), mask, mask_qs, q_target, phasec);
}

__global__ 
void kernel_suqa_swap(double *const state_re, double *const state_im, uint len, uint mask00, uint mask11, uint mask_q1, uint mask_q2){
    int i = blockDim.x*blockIdx.x + threadIdx.x;    
    while(i<len){
        if((i & mask11) == mask00){
            // i -> ...00..., i_1 -> ...10..., i_2 -> ...01...
            uint i_1 = i | mask_q1;
            uint i_2 = i | mask_q2;
            double tmpval = state_re[i_1];
            state_re[i_1]=state_re[i_2];
            state_re[i_2]=tmpval;
            tmpval = state_im[i_1];
            state_im[i_1]=state_im[i_2];
            state_im[i_2]=tmpval;
        }
        i+=gridDim.x*blockDim.x;
    }
}

void suqa::apply_swap(ComplexVec& state, const uint& q1, const uint& q2){
    // swap gate: 00->00, 01->10, 10->01, 11->11
    // equivalent to cx(q1,q2)->cx(q2,q1)->cx(q1,q2)
    uint mask00 = gc_mask;
    uint mask11 = mask00;
    uint mask_q1 = (1U << q1);
    uint mask_q2 = (1U << q2);
    mask11 |= mask_q1 | mask_q2;
    kernel_suqa_swap<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), mask00, mask11, mask_q1, mask_q2);
}

// RESET = measure + classical cx

// sets amplitudes with value <val> in qubit <q> to zero
// !! it leaves the state unnormalized !!
__global__ void kernel_suqa_set_ampl_to_zero(double *state_re, double *state_im, uint len, uint q, uint val){
    uint i =  blockIdx.x*blockDim.x + threadIdx.x;
    while(i<len){
        if(((i >> q) & 1U) == val){
            state_re[i] = 0.0;
            state_im[i] = 0.0;
        }
        i += gridDim.x*blockDim.x;
    }
}

__global__
void kernel_suqa_pauli_TP_rotation_x(double *const state_re, double *const state_im, uint len, uint mask0s, uint mask1s, uint mask_q1, double ctheta, double stheta){
    int i_0 = blockDim.x*blockIdx.x + threadIdx.x;
    double tmp_re0, tmp_im0, tmp_re1, tmp_im1;
    while(i_0<len){
        if((i_0 & mask1s) == mask0s){
            // i -> ...00..., i_1 -> ...01..., i_2 -> ...10...
            uint i_1 = i_0 | mask_q1;


            tmp_re0 = state_re[i_0];
            tmp_im0 = state_im[i_0];
            tmp_re1 = state_re[i_1];
            tmp_im1 = state_im[i_1];
            
            state_re[i_0] = tmp_re0*ctheta - tmp_im1*stheta;
            state_im[i_0] = tmp_im0*ctheta + tmp_re1*stheta;

            state_re[i_1] = tmp_re1*ctheta - tmp_im0*stheta; 
            state_im[i_1] = tmp_im1*ctheta + tmp_re0*stheta;
        }
        i_0+=gridDim.x*blockDim.x;
    }
}

__global__
void kernel_suqa_pauli_TP_rotation_y(double *const state_re, double *const state_im, uint len, uint mask0s, uint mask1s, uint mask_q1, double ctheta, double stheta){
    int i_0 = blockDim.x*blockIdx.x + threadIdx.x;
    double tmp_re0, tmp_im0, tmp_re1, tmp_im1;
    while(i_0<len){
        if((i_0 & mask1s) == mask0s){
            // i -> ...00..., i_1 -> ...01..., i_2 -> ...10...
            uint i_1 = i_0 | mask_q1;


            tmp_re0 = state_re[i_0];
            tmp_im0 = state_im[i_0];
            tmp_re1 = state_re[i_1];
            tmp_im1 = state_im[i_1];
            
            state_re[i_0] = tmp_re0*ctheta + tmp_re1*stheta;
            state_im[i_0] = tmp_im0*ctheta + tmp_im1*stheta;

            state_re[i_1] = tmp_re1*ctheta - tmp_re0*stheta; 
            state_im[i_1] = tmp_im1*ctheta - tmp_im0*stheta;
        }
        i_0+=gridDim.x*blockDim.x;
    }
}

__global__
void kernel_suqa_pauli_TP_rotation_z(double *const state_re, double *const state_im, uint len, uint mask0s, uint mask1s, uint mask_q1, double ctheta, double stheta){
    int i_0 = blockDim.x*blockIdx.x + threadIdx.x;
    double tmp_re0, tmp_im0, tmp_re1, tmp_im1;
    while(i_0<len){
        if((i_0 & mask1s) == mask0s){
            // i -> ...00..., i_1 -> ...01..., i_2 -> ...10...
            uint i_1 = i_0 | mask_q1;


            tmp_re0 = state_re[i_0];
            tmp_im0 = state_im[i_0];
            tmp_re1 = state_re[i_1];
            tmp_im1 = state_im[i_1];
            
            state_re[i_0] = tmp_re0*ctheta - tmp_im0*stheta;
            state_im[i_0] = tmp_im0*ctheta + tmp_re0*stheta;

            state_re[i_1] = tmp_re1*ctheta + tmp_im1*stheta; 
            state_im[i_1] = tmp_im1*ctheta - tmp_re1*stheta;
        }
        i_0+=gridDim.x*blockDim.x;
    }
}

__global__
void kernel_suqa_pauli_TP_rotation_zxx(double *const state_re, double *const state_im, uint len, uint mask0s, uint mask1s, uint mask_q1, uint mask_q2, uint mask_q3, double ctheta, double stheta){
    int i_0 = blockDim.x*blockIdx.x + threadIdx.x;
    double tmpval;
    while(i_0<len){
        if((i_0 & mask1s) == mask0s){
            // i -> ...00..., i_1 -> ...01..., i_2 -> ...10...
            uint i_1 = i_0 | mask_q1;
            uint i_2 = i_0 | mask_q2;
            uint i_3 = i_2 | i_1;
            uint i_4 = i_0 | mask_q3;
            uint i_5 = i_4 | i_1;
            uint i_6 = i_4 | i_2;
            uint i_7 = i_4 | i_3;
            
            // 0<->3
            tmpval = state_re[i_0];
            state_re[i_0] = tmpval*ctheta -state_im[i_3]*stheta;
            state_im[i_3] = state_im[i_3]*ctheta +tmpval*stheta;

            tmpval = state_im[i_0]; state_im[i_0] = tmpval*ctheta +state_re[i_3]*stheta;
            state_re[i_3] = state_re[i_3]*ctheta -tmpval*stheta;

            // 5<->6
            tmpval = state_re[i_5];
            state_re[i_5] = tmpval*ctheta +state_im[i_6]*stheta;
            state_im[i_6] = state_im[i_6]*ctheta -tmpval*stheta;

            tmpval = state_im[i_5];
            state_im[i_5] = tmpval*ctheta -state_re[i_6]*stheta;
            state_re[i_6] = state_re[i_6]*ctheta +tmpval*stheta;

            // 1<->2
            tmpval = state_re[i_1];
            state_re[i_1] = tmpval*ctheta -state_im[i_2]*stheta;
            state_im[i_2] = state_im[i_2]*ctheta +tmpval*stheta;

            tmpval = state_im[i_1];
            state_im[i_1] = tmpval*ctheta +state_re[i_2]*stheta;
            state_re[i_2] = state_re[i_2]*ctheta -tmpval*stheta;


            // 4<->7
            tmpval = state_re[i_4];
            state_re[i_4] = tmpval*ctheta +state_im[i_7]*stheta;
            state_im[i_7] = state_im[i_7]*ctheta -tmpval*stheta;

            tmpval = state_im[i_4];
            state_im[i_4] = tmpval*ctheta -state_re[i_7]*stheta;
            state_re[i_7] = state_re[i_7]*ctheta +tmpval*stheta;

        }
        i_0+=gridDim.x*blockDim.x;
    }
}

__global__
void kernel_suqa_pauli_TP_rotation_zyy(double *const state_re, double *const state_im, uint len, uint mask0s, uint mask1s, uint mask_q1, uint mask_q2, uint mask_q3, double ctheta, double stheta){
    int i_0 = blockDim.x*blockIdx.x + threadIdx.x;
    double tmpval;
    while(i_0<len){
        if((i_0 & mask1s) == mask0s){
            // i -> ...00..., i_1 -> ...01..., i_2 -> ...10...
            uint i_1 = i_0 | mask_q1;
            uint i_2 = i_0 | mask_q2;
            uint i_3 = i_2 | i_1;
            uint i_4 = i_0 | mask_q3;
            uint i_5 = i_4 | i_1;
            uint i_6 = i_4 | i_2;
            uint i_7 = i_4 | i_3;

            
            // 0<->3
            tmpval = state_re[i_0];
            state_re[i_0] = tmpval*ctheta +state_im[i_3]*stheta;
            state_im[i_3] = state_im[i_3]*ctheta -tmpval*stheta;

            tmpval = state_im[i_0];
            state_im[i_0] = tmpval*ctheta -state_re[i_3]*stheta;
            state_re[i_3] = state_re[i_3]*ctheta +tmpval*stheta;

            // 5<->6
            tmpval = state_re[i_5];
            state_re[i_5] = tmpval*ctheta +state_im[i_6]*stheta;
            state_im[i_6] = state_im[i_6]*ctheta -tmpval*stheta;

            tmpval = state_im[i_5];
            state_im[i_5] = tmpval*ctheta -state_re[i_6]*stheta;
            state_re[i_6] = state_re[i_6]*ctheta +tmpval*stheta;

            // 1<->2
            tmpval = state_re[i_1];
            state_re[i_1] = tmpval*ctheta -state_im[i_2]*stheta;
            state_im[i_2] = state_im[i_2]*ctheta +tmpval*stheta;

            tmpval = state_im[i_1];
            state_im[i_1] = tmpval*ctheta +state_re[i_2]*stheta;
            state_re[i_2] = state_re[i_2]*ctheta -tmpval*stheta;

            // 4<->7
            tmpval = state_re[i_4];
            state_re[i_4] = tmpval*ctheta -state_im[i_7]*stheta;
            state_im[i_7] = state_im[i_7]*ctheta +tmpval*stheta;

            tmpval = state_im[i_4];
            state_im[i_4] = tmpval*ctheta +state_re[i_7]*stheta;
            state_re[i_7] = state_re[i_7]*ctheta -tmpval*stheta;

        }
        i_0+=gridDim.x*blockDim.x;
    }
}


// rotation by phase in the direction of a pauli tensor product
void suqa::apply_pauli_TP_rotation(ComplexVec& state, const bmReg& q_apply, const std::vector<uint>& pauli_TPtype, double phase){
    uint mask0s = gc_mask;
    uint mask1s = mask0s;
    uint mask_q1, mask_q2, mask_q3;
    for(const auto& q : q_apply){
        mask1s |= (1U << q);
    }
    double sph, cph;
    sincos(phase, &sph, &cph);

    if(q_apply.size()!=pauli_TPtype.size()){
        throw std::runtime_error("ERROR: in suqa::apply_pauli_TP_rotation(): mismatch between qubits number and pauli types specified");
    }

    if(q_apply.size()==1U){
        mask_q1 = (1U << q_apply[0]);
        switch(pauli_TPtype[0]){
            case PAULI_X:
                kernel_suqa_pauli_TP_rotation_x<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), mask0s, mask1s, mask_q1, cph, sph);
                break;
            case PAULI_Y:
                kernel_suqa_pauli_TP_rotation_y<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), mask0s, mask1s, mask_q1, cph, sph);
                break;
            case PAULI_Z:
                kernel_suqa_pauli_TP_rotation_z<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), mask0s, mask1s, mask_q1, cph, sph);
                break;
            default:
                break;
        }
    }else if(q_apply.size()==3U){
        int i_z = -1, i1, i2;
        if(pauli_TPtype[0]==PAULI_Z){ 
            i_z=0;
            i1=1;
            i2=2;
        }else if(pauli_TPtype[1]==PAULI_Z){
            i_z=1;
            i1=0;
            i2=2;
        }else if(pauli_TPtype[2]==PAULI_Z){
            i_z=2;
            i1=0;
            i2=1;
        }else{
            throw std::runtime_error("ERROR: unimplemented pauli TP rotation with 3 qubits in the selected configuration");
        }
        mask_q3 = (1U << q_apply[i_z]);
        mask_q1 = (1U << q_apply[i1]);
        mask_q2 = (1U << q_apply[i2]);
        if(pauli_TPtype[i1]==PAULI_X and pauli_TPtype[i2]==PAULI_X){
                kernel_suqa_pauli_TP_rotation_zxx<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), mask0s, mask1s, mask_q1, mask_q2, mask_q3, cph, sph);
        }else if(pauli_TPtype[i1]==PAULI_Y and pauli_TPtype[i2]==PAULI_Y){
                kernel_suqa_pauli_TP_rotation_zyy<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im, state.size(), mask0s, mask1s, mask_q1, mask_q2, mask_q3, cph, sph);
        }else{
            throw std::runtime_error("ERROR: unimplemented pauli TP rotation with 3 qubits in the selected configuration");
        }
    }else{
        throw std::runtime_error(("ERROR: unimplemented pauli tensor product rotation with "+std::to_string(q_apply.size())+" qubits").c_str());
    }
}




void set_ampl_to_zero(ComplexVec& state, const uint& q, const uint& val){
    kernel_suqa_set_ampl_to_zero<<<suqa::blocks, suqa::threads>>>(state.data_re, state.data_im, state.size(), q, val);
}

__global__ void kernel_suqa_prob1(double *dev_partial_ret_ptr, double *v_re, double *v_im, uint len, uint q){
    extern __shared__ double local_ret[];
    uint tid = threadIdx.x;
    uint i =  blockIdx.x*blockDim.x + threadIdx.x;

//    double vj = v_comp[i+(blockDim.x >> 1)];
//    local_ret[tid] =  v_comp[i]*v_comp[i]+vj*vj;
    local_ret[threadIdx.x] = 0.0;
    double tmpval;
    while(i<len){
        if(i & (1U << q)){
            tmpval = v_re[i];
            local_ret[tid] +=  tmpval*tmpval;
            tmpval = v_im[i];
            local_ret[tid] +=  tmpval*tmpval;
        }
        i += gridDim.x*blockDim.x;
//        printf("v[%d] = (%.16lg, %.16lg)\n",i, v_re[i], v_im[i]);
//        printf("local_ret[%d] = %.10lg\n",tid, local_ret[tid]);

    }
    __syncthreads();

    for(uint s=blockDim.x/2; s>0; s>>=1){
        if(tid < s){
            local_ret[tid] += local_ret[tid+s];
        }
        __syncthreads();
    }
//    if (blockDim.x >= 1024) { if (tid < 512) { local_ret[tid] += local_ret[tid + 512]; } __syncthreads(); }
//    if (blockDim.x >=  512) { if (tid < 256) { local_ret[tid] += local_ret[tid + 256]; } __syncthreads(); }
//    if (blockDim.x >=  256) { if (tid < 128) { local_ret[tid] += local_ret[tid + 128]; } __syncthreads(); }
//    if (blockDim.x >=  128) { if (tid <  64) { local_ret[tid] += local_ret[tid +  64]; } __syncthreads(); }
//
//    if(tid<32){
//        if (blockDim.x >= 64) local_ret[tid] += local_ret[tid + 32];
//        if (blockDim.x >= 32) local_ret[tid] += local_ret[tid + 16];
//        if (blockDim.x >= 16) local_ret[tid] += local_ret[tid +  8];
//        if (blockDim.x >=  8) local_ret[tid] += local_ret[tid +  4];
//        if (blockDim.x >=  4) local_ret[tid] += local_ret[tid +  2];
//        if (blockDim.x >=  2) local_ret[tid] += local_ret[tid +  1];
//    }

    if(tid==0) dev_partial_ret_ptr[blockIdx.x] = local_ret[0];
}


void suqa::measure_qbit(ComplexVec& state, uint q, uint& c, double rdoub){
    double prob1 = 0.0;
    c=0U;
//    kernel_suqa_prob1<<<suqa::blocks,suqa::threads,suqa::threads*sizeof(double)>>>(dev_partial_ret, state.data, 2*state.size(), q);
    kernel_suqa_prob1<<<suqa::blocks,suqa::threads,suqa::threads*sizeof(double)>>>(dev_partial_ret, state.data_re, state.data_im, state.size(), q);
//    kernel_suqa_prob1<<<suqa::blocks,suqa::threads,suqa::threads*sizeof(double),stream2>>>(dev_partial_ret+blocks, state.data_im, state.size(), q);
//    hipDeviceSynchronize();
    hipMemcpy(host_partial_ret,dev_partial_ret,suqa::blocks*sizeof(double), hipMemcpyDeviceToHost);
    
    for(uint bid=0; bid<suqa::blocks && prob1<rdoub; ++bid){
        prob1 += host_partial_ret[bid]; 
    } 

    c = (uint)(rdoub < prob1); // prob1=1 -> c = 1 surely
    uint c_conj = c^1U; // 1U-c, since c=0U or 1U


#ifndef NDEBUG
    std::cout<<"prob1="<<prob1<<", c_conj="<<c_conj<<std::endl;
#endif
//#if defined(CUDA_HOST)
//    DEBUG_CALL(std::cout<<"before flipping qbit "<<c_conj<<std::endl);
//    DEBUG_CALL(sparse_print((double*)state.data, state.size()));
//#endif

    // set to 0 coeffs with bm_acc 1-c
    set_ampl_to_zero(state, q, c_conj);
    suqa::vnormalize(state);
}

////TODO: can be optimized for multiple qbits measures?
void suqa::measure_qbits(ComplexVec& state, const bmReg& qs, std::vector<uint>& cs,const std::vector<double>& rdoubs){
    for(uint k = 0U; k < qs.size(); ++k)
        suqa::measure_qbit(state, qs[k], cs[k], rdoubs[k]);
}


__global__ void kernel_suqa_prob_filter(double *dev_partial_ret_ptr, double *v_re, double *v_im, uint len, uint mask_qs, uint mask){
    extern __shared__ double local_ret[];
    uint tid = threadIdx.x;
    uint i =  blockIdx.x*blockDim.x + threadIdx.x;

//    double vj = v_comp[i+(blockDim.x >> 1)];
//    local_ret[tid] =  v_comp[i]*v_comp[i]+vj*vj;
    local_ret[threadIdx.x] = 0.0;
    double tmpval;
    while(i<len){
        if((i & mask_qs) == mask){
            tmpval = v_re[i];
            local_ret[tid] +=  tmpval*tmpval;
            tmpval = v_im[i];
            local_ret[tid] +=  tmpval*tmpval;
        }
        i += gridDim.x*blockDim.x;
//        printf("v[%d] = (%.16lg, %.16lg)\n",i, v_re[i], v_im[i]);
//        printf("local_ret[%d] = %.10lg\n",tid, local_ret[tid]);

    }
    __syncthreads();

    for(uint s=blockDim.x/2; s>0; s>>=1){
        if(tid < s){
            local_ret[tid] += local_ret[tid+s];
        }
        __syncthreads();
    }
//    if (blockDim.x >= 1024) { if (tid < 512) { local_ret[tid] += local_ret[tid + 512]; } __syncthreads(); }
//    if (blockDim.x >=  512) { if (tid < 256) { local_ret[tid] += local_ret[tid + 256]; } __syncthreads(); }
//    if (blockDim.x >=  256) { if (tid < 128) { local_ret[tid] += local_ret[tid + 128]; } __syncthreads(); }
//    if (blockDim.x >=  128) { if (tid <  64) { local_ret[tid] += local_ret[tid +  64]; } __syncthreads(); }
//
//    if(tid<32){
//        if (blockDim.x >= 64) local_ret[tid] += local_ret[tid + 32];
//        if (blockDim.x >= 32) local_ret[tid] += local_ret[tid + 16];
//        if (blockDim.x >= 16) local_ret[tid] += local_ret[tid +  8];
//        if (blockDim.x >=  8) local_ret[tid] += local_ret[tid +  4];
//        if (blockDim.x >=  4) local_ret[tid] += local_ret[tid +  2];
//        if (blockDim.x >=  2) local_ret[tid] += local_ret[tid +  1];
//    }

    if(tid==0) dev_partial_ret_ptr[blockIdx.x] = local_ret[0];
}



void suqa::prob_filter(ComplexVec& state, const bmReg& qs, const std::vector<uint>& q_mask, double &prob){
    prob = 0.0;
    uint mask_qs = 0U;
    for(const auto& q : qs)
        mask_qs |= 1U << q;
    uint mask = 0U;
    for(uint k = 0U; k < q_mask.size(); ++k){
        if(q_mask[k]) mask |= q_mask[k] << qs[k];
    }
//    kernel_suqa_prob_filter<<<suqa::blocks,suqa::threads,suqa::threads*sizeof(double)>>>(dev_partial_ret, state.data, 2*state.size(), q);
    kernel_suqa_prob_filter<<<suqa::blocks,suqa::threads,suqa::threads*sizeof(double)>>>(dev_partial_ret, state.data_re, state.data_im, state.size(), mask_qs, mask);
//    kernel_suqa_prob_filter<<<suqa::blocks,suqa::threads,suqa::threads*sizeof(double)>>>(dev_partial_ret+suqa::blocks, state.data_im, state.size(), mask_qs, mask);
//    hipDeviceSynchronize();
    hipMemcpy(host_partial_ret,dev_partial_ret,suqa::blocks*sizeof(double), hipMemcpyDeviceToHost);
    
    for(uint bid=0; bid<suqa::blocks; ++bid){
        prob += host_partial_ret[bid]; 
    } 
}


void suqa::apply_reset(ComplexVec& state, uint q, double rdoub){
//    DEBUG_CALL(std::cout<<"Calling apply_reset() with q="<<q<<"and rdoub="<<rdoub<<std::endl);
    uint c;
    suqa::measure_qbit(state, q, c, rdoub);
    if(c){ // c==1U
        suqa::apply_x(state, q);
        // suqa::vnormalize(state); // normalization shoud be guaranteed by the measure
    }
}  

// fake reset
//void suqa::apply_reset(ComplexVec& state, uint q, double rdoub){
//    for(uint i = 0U; i < state.size(); ++i){
//        if((i >> q) & 1U){ // checks q-th digit in i
//            uint j = i & ~(1U << q); // j has 0 on q-th digit
//            state[j]+=state[i];
//            state[i].x = 0.0;
//            state[i].y = 0.0;
//        }
//    }
//}

void suqa::apply_reset(ComplexVec& state, const bmReg& qs, std::vector<double> rdoubs){
    // qs.size() == rdoubs.size()
    for(uint i=0; i<qs.size(); ++i){
        suqa::apply_reset(state, qs[i], rdoubs[i]); 
    } 
}

void suqa::setup(uint Dim){
    hipHostAlloc((void**)&host_partial_ret,suqa::blocks*sizeof(double),hipHostMallocDefault);
    hipMalloc((void**)&dev_partial_ret, suqa::blocks*sizeof(double));  
// the following are allocated only for library versions of reduce
//    hipHostAlloc((void**)&ret_re_im,2*sizeof(double),hipHostMallocDefault);
//    hipMalloc((void**)&d_ret_re_im,2*sizeof(double));

    hipDeviceProp_t prop;
    int whichDevice;
    HANDLE_CUDACALL( hipGetDevice( &whichDevice ) );
    HANDLE_CUDACALL( hipGetDeviceProperties( &prop, whichDevice ) );

    HANDLE_CUDACALL( hipStreamCreate( &suqa::stream1 ) );
    if (!prop.deviceOverlap) {
        DEBUG_CALL(printf( "Device will not handle overlaps, so no "
        "speed up from streams\n" ));
        suqa::stream2 = suqa::stream1;
    }else{
        HANDLE_CUDACALL( hipStreamCreate( &suqa::stream2 ) );
    }

#if !defined(NDEBUG)
    HANDLE_CUDACALL(hipHostAlloc((void**)&host_state_re,Dim*sizeof(double),hipHostMallocDefault));
    HANDLE_CUDACALL(hipHostAlloc((void**)&host_state_im,Dim*sizeof(double),hipHostMallocDefault));
#endif
}

void suqa::clear(){
//    hipFree(d_ret_re_im);
//    hipHostFree(ret_re_im);
    hipFree(dev_partial_ret); 
    hipHostFree(host_partial_ret);

#ifndef NDEBUG
    HANDLE_CUDACALL(hipHostFree(host_state_re));
    HANDLE_CUDACALL(hipHostFree(host_state_im));
#endif

    HANDLE_CUDACALL( hipStreamDestroy( suqa::stream1 ) );
    if (suqa::stream1!=suqa::stream2) {
        HANDLE_CUDACALL( hipStreamDestroy( suqa::stream2 ) );
    }

}
