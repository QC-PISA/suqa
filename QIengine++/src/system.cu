#include "hip/hip_runtime.h"
#include "system.cuh"
#include "Rand.hpp"



//TODO: make the number of "state" qubits determined at compilation time in system.cuh
double g_beta;


__global__ void initialize_state(double *state_re, double *state_im, uint len){
    uint i = blockIdx.x*blockDim.x+threadIdx.x;
    while(i<len){
        state_re[i] = 0.0;
        state_im[i] = 0.0;
        i += gridDim.x*blockDim.x;
    }
    if(blockIdx.x*blockDim.x+threadIdx.x==1){
        state_re[0] = 1.0;
        state_im[0] = 0.0;
    }
}




void init_state(ComplexVec& state, uint Dim){

    if(state.size()!=Dim)
	throw std::runtime_error("ERROR: init_state() failed");
    

    initialize_state<<<suqa::blocks,suqa::threads>>>(state.data_re, state.data_im,Dim);

	suqa::apply_x(state, bm_spin[1]);
	suqa::apply_h(state, bm_spin[1]);
	suqa::apply_cx(state, bm_spin[1], bm_spin[0]);
}




void exp_it_id_x_x( ComplexVec& state, const bmReg& q, uint pos_id, double phase_t){
	
	suqa::apply_pauli_TP_rotation(state, {q[(pos_id+1)%3],q[(pos_id+2)%3]}, {PAULI_X,PAULI_X}, phase_t);

}

void evolution(ComplexVec& state, const double& t, const int& n){

	for (uint iii=0; iii<3; ++iii){
		exp_it_id_x_x(state, bm_spin, iii, -t);
 	}

}


/* Measure facilities */
const uint op_bits = 3; // 2^op_bits is the number of eigenvalues for the observable
const bmReg bm_op = bm_spin; // where the measure has to be taken
const std::vector<double> op_vals = {2.0,0.0,2.0,0.0,-2.0,0.0,-2.0,0.0}; // eigvals

 
// change basis to the observable basis somewhere in the system registers
void apply_measure_rotation(ComplexVec& state){
	suqa::apply_h(state, bm_spin[0]);
	suqa::apply_h(state, bm_spin[1]);
	suqa::apply_h(state, bm_spin[2]);
	suqa::apply_cx(state, bm_spin[0], bm_spin[1]);
	suqa::apply_cx(state, bm_spin[0], bm_spin[2]);
	suqa::apply_u1(state,bm_spin[2], M_PI*0.5);
}

// inverse of the above function
void apply_measure_antirotation(ComplexVec& state){
 	apply_measure_rotation(state);
}

// map the classical measure recorded in creg_vals
// to the corresponding value of the observable;
// there is no need to change it
double get_meas_opvals(const uint& creg_vals){
    return op_vals[creg_vals];
}

// actually perform the measure
// there is no need to change it
double measure_X(ComplexVec& state, pcg& rgen){
    std::vector<uint> classics(op_bits);
    
    apply_measure_rotation(state);

    std::vector<double> rdoubs(op_bits);
    for(auto& el : rdoubs){
        el = rgen.doub();
    }
    suqa::measure_qbits(state, bm_op, classics, rdoubs);

    apply_measure_antirotation(state);

    uint meas = 0U;
    for(uint i=0; i<op_bits; ++i){
        meas |= (classics[i] << i);
    }

    return get_meas_opvals(meas);
    return 0.0;
}

/* Moves facilities */

std::vector<double> C_weigthsums = {1./3, 2./3, 1.0};




void apply_C(ComplexVec& state, const uint &Ci){
    if(Ci>2)
        throw std::runtime_error("ERROR: wrong move selection");

    suqa::apply_h(state,bm_spin[Ci]);
}


void apply_C_inverse(ComplexVec& state, const uint &Ci){
    apply_C(state, Ci);
}

std::vector<double> get_C_weigthsums(){ return C_weigthsums; }

